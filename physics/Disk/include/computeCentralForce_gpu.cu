#include "hip/hip_runtime.h"
//
// Created by Noah Kubli on 11.03.2024.
//
#include <hipcub/hipcub.hpp>
#include "hip/hip_runtime.h"

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/primitives/math.hpp"
#include "cstone/sfc/box.hpp"
#include "cstone/traversal/find_neighbors.cuh"
#include "sph/particles_data.hpp"

#include "computeCentralForce_gpu.hpp"
#include "star_data.hpp"

namespace disk
{

template<typename T>
__device__ void atomicAddVec4(cstone::Vec4<T>* x, const cstone::Vec4<T>& y)
{
    atomicAdd(&(*x)[0], y[0]);
    atomicAdd(&(*x)[1], y[1]);
    atomicAdd(&(*x)[2], y[2]);
    atomicAdd(&(*x)[3], y[3]);
}

template<size_t numThreads, typename Tpos, typename Ta, typename Tsp>
__global__ void computeCentralForceGPUKernel(size_t first, size_t last, const Tpos* x, const Tpos* y, const Tpos* z,
                                             Ta* ax, Ta* ay, Ta* az, const auto* m, const cstone::Vec3<Tsp> star_position,
                                             auto sm, auto g, auto inner_size2, auto* force_device)
{
    cstone::LocalIndex i = first + blockDim.x * blockIdx.x + threadIdx.x;
    Tf                 force{};

    if (i >= last) { force = {0., 0., 0., 0.}; }
    else
    {
        const double dx    = x[i] - star_position[0];
        const double dy    = y[i] - star_position[1];
        const double dz    = z[i] - star_position[2];
        const double dist2 = stl::max(inner_size2, dx * dx + dy * dy + dz * dz);
        const double dist  = sqrt(dist2);
        const double dist3 = dist2 * dist;

        const double a_strength = 1. / dist3 * sm * g;
        const double ax_i       = -dx * a_strength;
        const double ay_i       = -dy * a_strength;
        const double az_i       = -dz * a_strength;
        ax[i] += ax_i;
        ay[i] += ay_i;
        az[i] += az_i;

        force[0] = -g * m[i] / dist;
        force[1] = -ax_i * m[i];
        force[2] = -ay_i * m[i];
        force[3] = -az_i * m[i];
    }

    typedef hipcub::BlockReduce<Tf, numThreads>     BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    Tf force_block = BlockReduce(temp_storage).Sum(force);
    __syncthreads();
    if (threadIdx.x == 0) { atomicAddVec4(force_device, force_block); }
}

template<typename Treal, typename Tmass>
void computeCentralForceGPU(size_t first, size_t last, const Treal* x, const Treal* y, const Treal* z, Treal* ax,
                            Treal* ay, Treal* az, const Tmass* m, StarData& star)
{
    cstone::LocalIndex numParticles = last - first;
    constexpr unsigned numThreads   = 256;
    unsigned           numBlocks    = (numParticles + numThreads - 1) / numThreads;

    star.force_local = {};
    cstone::Vec4<double>* force_device;
    checkGpuErrors(hipMalloc(reinterpret_cast<void**>(&force_device), sizeof *force_device));
    checkGpuErrors(hipMemcpy(force_device, &star.force_local, sizeof star.force_local, hipMemcpyHostToDevice));

    computeCentralForceGPUKernel<numThreads>
        <<<numBlocks, numThreads>>>(first, last, x, y, z, ax, ay, az, m, star.position, star.m, d.g,
                                    star.inner_size * star.inner_size, force_device);

    checkGpuErrors(hipDeviceSynchronize());
    checkGpuErrors(hipGetLastError());

    checkGpuErrors(hipMemcpy(&star.force_local, force_device, sizeof star.force_local, hipMemcpyDeviceToHost));
    checkGpuErrors(hipFree(force_device));
}

#define COMPUTE_CENTRAL_FORCE_GPU(Treal, Tmass)                                                                        \
    template void computeCentralForceGPU(size_t, size_t, const Treal* x, const Treal* y, const Treal* z, Treal* ax,    \
                                         Treal* ay, Treal* az, const Tmass* m, StarData&);

COMPUTE_CENTRAL_FORCE_GPU(double, double);
COMPUTE_CENTRAL_FORCE_GPU(double, float);

} // namespace disk
