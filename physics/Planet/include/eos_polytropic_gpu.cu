#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Density i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

//#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/primitives/math.hpp"
#include "cstone/util/tuple.hpp"
#include "eos_polytropic_gpu.hpp"
#include "eos_polytropic_loop.hpp"
#include "sph/particles_data.hpp"
#include "star_data.hpp"

namespace planet
{

template<typename T1, typename T2, typename T3, typename Trho, typename Tp, typename Tc>
__global__ void computePolytropicEOS_HydroStdKernel(size_t firstParticle, size_t lastParticle, T1 Kpoly, T2 exp_poly,
                                                    T3 gamma, const Trho* rho, Tp* p, Tc* c)
{
    unsigned i = firstParticle + blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= lastParticle) return;

    util::tie(p[i], c[i]) = polytropicEOS(Kpoly, exp_poly, gamma, rho[i]);
}

template<typename Dataset, typename StarData>
void computePolytropicEOS_HydroStdGPU(size_t firstParticle, size_t lastParticle, Dataset& d, const StarData& star)
{
    if (firstParticle == lastParticle) { return; }
    unsigned numThreads = 256;
    unsigned numBlocks  = cstone::iceil(lastParticle - firstParticle, numThreads);
    computePolytropicEOS_HydroStdKernel<<<numBlocks, numThreads>>>(firstParticle, lastParticle, star.Kpoly,
                                                                   star.exp_poly, d.gamma, rawPtr(d.devData.rho),
                                                                   rawPtr(d.devData.p), rawPtr(d.devData.c));

    checkGpuErrors(hipDeviceSynchronize());
}

template void computePolytropicEOS_HydroStdGPU(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, const StarData&);
} // namespace planet
