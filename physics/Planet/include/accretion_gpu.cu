#include "hip/hip_runtime.h"
//
// Created by Noah Kubli on 12.03.2024.
//
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/sequence.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"
#include "sph/util/device_math.cuh"

#include "cstone/sfc/box.hpp"
#include "cstone/tree/definitions.h"

#include "sph/particles_data.hpp"

#include "accretion_gpu.hpp"
#include "star_data.hpp"
#include "hip/hip_runtime.h"

struct RemovalProperties
{
    double   mass        = 0.;
    double   momentum[3] = {0.};
    unsigned count       = 0;
    //    RemovalProperties operator+(const RemovalProperties& b) const
    //    {
    //        return RemovalProperties{mass + b.mass,
    //                                 {
    //                                     momentum[0] + b.momentum[0],
    //                                     momentum[1] + b.momentum[1],
    //                                     momentum[2] + b.momentum[2],
    //                                 },
    //                                 count + b.count};
    //    }
};

__device__ void atomicAdd(RemovalProperties* x, const RemovalProperties& y)
{
    atomicAdd(&(x->mass), y.mass);
    atomicAdd(&(x->momentum[0]), y.momentum[0]);
    atomicAdd(&(x->momentum[1]), y.momentum[1]);
    atomicAdd(&(x->momentum[2]), y.momentum[2]);
    atomicAdd(&(x->count), y.count);
}

static __device__ RemovalProperties device_accreted;
static __device__ RemovalProperties device_removed;

using cstone::TravConfig;

template<typename T1, typename Th, typename Tkeys, typename T2, typename Tm, typename Tv>
__global__ void computeAccretionConditionKernel(size_t first, size_t last, const T1* x, const T1* y, const T1* z,
                                                const Th* h, Tkeys* keys, const Tm* m, const Tv* vx, const Tv* vy,
                                                const Tv* vz, T2 star_x, T2 star_y, T2 star_z, T2 star_size2,
                                                T2 removal_limit_h)
{
    cstone::LocalIndex i = first + blockDim.x * blockIdx.x + threadIdx.x;

    RemovalProperties accreted{};
    RemovalProperties removed{};

    if (i >= last) {}
    else
    {
        const double dx    = x[i] - star_x;
        const double dy    = y[i] - star_y;
        const double dz    = z[i] - star_z;
        const double dist2 = dx * dx + dy * dy + dz * dz;

        if (dist2 < star_size2)
        {
            // Accrete on star
            keys[i]              = cstone::removeKey<Tkeys>::value;
            accreted.mass        = m[i];
            accreted.momentum[0] = m[i] * vx[i];
            accreted.momentum[1] = m[i] * vy[i];
            accreted.momentum[2] = m[i] * vz[i];
            accreted.count       = 1;
        }
        else if (h[i] > removal_limit_h)
        {
            // Remove from system
            keys[i]             = cstone::removeKey<Tkeys>::value;
            removed.mass        = m[i];
            removed.momentum[0] = m[i] * vx[i];
            removed.momentum[1] = m[i] * vy[i];
            removed.momentum[2] = m[i] * vz[i];
            removed.count       = 1;
        }
    }

    typedef hipcub::BlockReduce<RemovalProperties, TravConfig::numThreads> BlockReduce;
    __shared__ typename BlockReduce::TempStorage                        temp_accreted, temp_removed;

    struct Sum
    {
        __device__ RemovalProperties operator()(const RemovalProperties& a, const RemovalProperties& b) const
        {
            return RemovalProperties{a.mass + b.mass,
                                     {
                                         a.momentum[0] + b.momentum[0],
                                         a.momentum[1] + b.momentum[1],
                                         a.momentum[2] + b.momentum[2],
                                     },
                                     a.count + b.count};
        }
    };

    RemovalProperties block_accreted = BlockReduce(temp_accreted).Reduce(accreted, Sum{});
    RemovalProperties block_removed  = BlockReduce(temp_removed).Reduce(removed, Sum{});

    __syncthreads();

    if (threadIdx.x == 0)
    {
        atomicAdd(&device_accreted, block_accreted);
        atomicAdd(&device_removed, block_removed);
    }
}

template<typename Dataset, typename StarData>
void computeAccretionConditionGPU(size_t first, size_t last, Dataset& d, StarData& star)
{
    cstone::LocalIndex numParticles = last - first;
    unsigned           numThreads   = 256;
    unsigned           numBlocks    = (numParticles + numThreads - 1) / numThreads;

    RemovalProperties accreted_local{}, removed_local{};

    hipMemcpyToSymbol(HIP_SYMBOL(device_accreted), &accreted_local, sizeof(accreted_local));
    hipMemcpyToSymbol(HIP_SYMBOL(device_removed), &removed_local, sizeof(removed_local));

    computeAccretionConditionKernel<<<numBlocks, numThreads>>>(
        first, last, rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.h),
        rawPtr(d.devData.keys), rawPtr(d.devData.m), rawPtr(d.devData.vx), rawPtr(d.devData.vy), rawPtr(d.devData.vz),
        star.position[0], star.position[1], star.position[2], star.inner_size * star.inner_size, star.removal_limit_h);
    checkGpuErrors(hipGetLastError());
    checkGpuErrors(hipDeviceSynchronize());

    hipMemcpyFromSymbol(&accreted_local, HIP_SYMBOL(device_accreted), sizeof(accreted_local));
    hipMemcpyFromSymbol(&removed_local, HIP_SYMBOL(device_removed), sizeof(removed_local));

    star.m_accreted_local    = accreted_local.mass;
    star.p_accreted_local[0] = accreted_local.momentum[0];
    star.p_accreted_local[1] = accreted_local.momentum[1];
    star.p_accreted_local[2] = accreted_local.momentum[2];
    star.n_accreted_local    = accreted_local.count;

    star.m_removed_local    = removed_local.mass;
    star.p_removed_local[0] = removed_local.momentum[0];
    star.p_removed_local[1] = removed_local.momentum[1];
    star.p_removed_local[2] = removed_local.momentum[2];
    star.n_removed_local    = removed_local.count;
}

template void computeAccretionConditionGPU(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, StarData&);
