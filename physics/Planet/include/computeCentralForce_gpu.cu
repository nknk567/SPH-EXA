#include "hip/hip_runtime.h"
//
// Created by Noah Kubli on 11.03.2024.
//
#include <hipcub/hipcub.hpp>
#include "hip/hip_runtime.h"

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/primitives/math.hpp"
#include "cstone/sfc/box.hpp"
#include "cstone/traversal/find_neighbors.cuh"
#include "sph/particles_data.hpp"
#include "sph/util/device_math.cuh"

#include "computeCentralForce_gpu.hpp"
#include "star_data.hpp"

namespace disk
{

template<typename T>
__device__ void atomicAddVec4(cstone::Vec4<T>* x, const cstone::Vec4<T>& y)
{
    atomicAdd(&(*x)[0], y[0]);
    atomicAdd(&(*x)[1], y[1]);
    atomicAdd(&(*x)[2], y[2]);
    atomicAdd(&(*x)[3], y[3]);
}

template<size_t numThreads, typename Tpos, typename Ta, typename Tm, typename Tsp, typename Tsm, typename Tg,
         typename Tis, typename Tf>
__global__ void computeCentralForceGPUKernel(size_t first, size_t last, const Tpos* x, const Tpos* y, const Tpos* z,
                                             Ta* ax, Ta* ay, Ta* az, const Tm* m, Tsp star_pos_x, Tsp star_pos_y,
                                             Tsp star_pos_z, Tsm sm, Tg g, Tis inner_size2, Tf* force_device)
{
    cstone::LocalIndex i = first + blockDim.x * blockIdx.x + threadIdx.x;
    Tf                 force{};

    if (i >= last) { force = {0., 0., 0., 0.}; }
    else
    {
        const double dx    = x[i] - star_pos_x;
        const double dy    = y[i] - star_pos_y;
        const double dz    = z[i] - star_pos_z;
        const double dist2 = stl::max(inner_size2, dx * dx + dy * dy + dz * dz);
        const double dist  = sqrt(dist2);
        const double dist3 = dist2 * dist;

        const double a_strength = 1. / dist3 * sm * g;
        const double ax_i       = -dx * a_strength;
        const double ay_i       = -dy * a_strength;
        const double az_i       = -dz * a_strength;
        ax[i] += ax_i;
        ay[i] += ay_i;
        az[i] += az_i;

        force[0] = -g * m[i] / dist;
        force[1] = -ax_i * m[i];
        force[2] = -ay_i * m[i];
        force[3] = -az_i * m[i];
    }

    typedef hipcub::BlockReduce<Tf, numThreads>     BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    Tf force_block = BlockReduce(temp_storage).Sum(force);
    __syncthreads();
    if (threadIdx.x == 0) { atomicAddVec4(force_device, force_block); }
}

template<typename Dataset, typename StarData>
void computeCentralForceGPU(size_t first, size_t last, Dataset& d, StarData& star)
{
    cstone::LocalIndex numParticles = last - first;
    constexpr unsigned numThreads   = 256;
    unsigned           numBlocks    = (numParticles + numThreads - 1) / numThreads;

    star.force_local = {};
    cstone::Vec4<double>* force_device;
    checkGpuErrors(hipMalloc(reinterpret_cast<void**>(&force_device), sizeof *force_device));
    checkGpuErrors(hipMemcpy(force_device, &star.force_local, sizeof star.force_local, hipMemcpyHostToDevice));

    computeCentralForceGPUKernel<numThreads><<<numBlocks, numThreads>>>(
        first, last, rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.ax),
        rawPtr(d.devData.ay), rawPtr(d.devData.az), rawPtr(d.devData.m), star.position[0], star.position[1],
        star.position[2], star.m, d.g, star.inner_size * star.inner_size, force_device);

    checkGpuErrors(hipDeviceSynchronize());
    checkGpuErrors(hipGetLastError());

    checkGpuErrors(hipMemcpy(&star.force_local, force_device, sizeof star.force_local, hipMemcpyDeviceToHost));
    checkGpuErrors(hipFree(force_device));
}

template void computeCentralForceGPU(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, StarData&);
} // namespace disk
