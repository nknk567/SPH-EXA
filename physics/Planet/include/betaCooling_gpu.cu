#include "hip/hip_runtime.h"
//
// Created by Noah Kubli on 17.04.2024.
//

#include "betaCooling_gpu.hpp"
#include "sph/util/device_math.cuh"
#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"
#include "cstone/sfc/box.hpp"

#include "sph/util/device_math.cuh"
#include "sph/particles_data.hpp"
#include "star_data.hpp"

template<typename Tpos, typename Tu, typename Ts, typename Tdu, typename Trho, typename Trho2>
__global__ void betaCoolingGPUKernel(size_t first, size_t last, const Tpos* x, const Tpos* y, const Tpos* z, Tdu* du,
                                     const Tu* u, Ts star_mass, Ts star_pos_x, Ts star_pos_y, Ts star_pos_z, Ts beta,
                                     Tpos g, const Trho* rho, Ts u_floor, Trho2 cooling_rho_limit)

{
    cstone::LocalIndex i = first + blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= last) { return; }
    if (rho[i] >= cooling_rho_limit || u[i] <= u_floor) return;

    const double dx    = x[i] - star_pos_x;
    const double dy    = y[i] - star_pos_y;
    const double dz    = z[i] - star_pos_z;
    const double dist2 = dx * dx + dy * dy + dz * dz;
    const double dist  = sqrt(dist2);
    const double omega = sqrt(g * star_mass / (dist2 * dist));
    du[i] += -u[i] * omega / beta;
}

template<typename Dataset, typename StarData>
void betaCoolingGPU(size_t first, size_t last, Dataset& d, StarData& star)
{
    cstone::LocalIndex numParticles = last - first;
    unsigned           numThreads   = 256;
    unsigned           numBlocks    = (numParticles + numThreads - 1) / numThreads;

    betaCoolingGPUKernel<<<numBlocks, numThreads>>>(
        first, last, rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.du),
        rawPtr(d.devData.u), star.m, star.position[0], star.position[1], star.position[2], star.beta, d.g,
        rawPtr(d.devData.rho), star.u_floor, star.cooling_rho_limit);

    checkGpuErrors(hipDeviceSynchronize());
}

template void betaCoolingGPU(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, const StarData&);

template<class T>
struct NormSquare3D
{
    HOST_DEVICE_FUN T operator()(const thrust::tuple<T, T, T>& X)
    {
        return get<0>(X) * get<0>(X) + get<1>(X) * get<1>(X) + get<2>(X) * get<2>(X);
    }
};

template<class T>
T maxNormSquareGpu(const T* x, const T* y, const T* z, size_t numElements)
{
    auto it1 = thrust::make_zip_iterator(x, y, z);
    auto it2 = thrust::make_zip_iterator(x + numElements, y + numElements, z + numElements);

    T init = 0;

    return thrust::transform_reduce(thrust::device, it1, it2, NormSquare3D<T>{}, init, thrust::maximum<T>{});
}

template<typename Tu, typename Tdu, typename Tt>
struct AbsDivide
{
    __global__ Tt operator() const(thrust::tuple<Tu, Tdu>& u_du) { return stl::abs(get<0>(u_du) / get<1>(u_du)); }
};

template<typename Dataset, typename StarData>
void duTimestep(size_t first, size_t last, const Dataset& d, const StarData& star)
{
    cstone::LocalIndex numParticles = last - first;

    const auto* u  = rawPtr(d.devData.u);
    const auto* du = rawPtr(d.devData.du);

    using Tu  = std::decay_t<decltype(*u)>;
    using Tdu = std::decay_t<decltype(*du)>;
    using Tt  = std::common_type_t<Tu, Tdu>;

    auto begin = thrust::make_zip_iterator(u, du);
    auto end   = thrust::make_zip_iterator(u + numParticles, du + numParticles);

    T init = INFINITY;

    return star.K_u *
           thrust::transform_reduce(thrust::device, it1, it2, AbsDivide<Tu, Tdu, Tt>{}, init, thrust::maximum<Tt>{});
}

template void duTimestep(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, const StarData&);
