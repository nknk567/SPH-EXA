#include "hip/hip_runtime.h"
//
// Created by Noah Kubli on 17.04.2024.
//

#include "betaCooling_gpu.hpp"
#include "sph/util/device_math.cuh"
#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"
#include "cstone/sfc/box.hpp"

#include "sph/util/device_math.cuh"
#include "sph/particles_data.hpp"
#include "star_data.hpp"

#include <thrust/reduce.h>
#include <thrust/tuple.h>

#include <cmath>

namespace disk
{

template<typename Tpos, typename Tu, typename Ts, typename Tdu, typename Trho, typename Trho2>
__global__ void betaCoolingGPUKernel(size_t first, size_t last, const Tpos* x, const Tpos* y, const Tpos* z, Tdu* du,
                                     const Tu* u, Ts star_mass, cstone::Vec3<Ts> star_position, Ts beta,
                                     Tpos g, const Trho* rho, Ts u_floor, Trho2 cooling_rho_limit)

{
    cstone::LocalIndex i = first + blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= last) { return; }
    if (rho[i] >= cooling_rho_limit || u[i] <= u_floor) return;

    const double dx    = x[i] - star_position[0];
    const double dy    = y[i] - star_position[1];
    const double dz    = z[i] - star_position[2];
    const double dist2 = dx * dx + dy * dy + dz * dz;
    const double dist  = sqrt(dist2);
    const double omega = sqrt(g * star_mass / (dist2 * dist));
    du[i] += -u[i] * omega / beta;
}

template<typename Dataset, typename StarData>
void betaCoolingGPU(size_t first, size_t last, Dataset& d, StarData& star)
{
    cstone::LocalIndex numParticles = last - first;
    unsigned           numThreads   = 256;
    unsigned           numBlocks    = (numParticles + numThreads - 1) / numThreads;

    betaCoolingGPUKernel<<<numBlocks, numThreads>>>(
        first, last, rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.du),
        rawPtr(d.devData.u), star.m, star.position, star.beta, d.g,
        rawPtr(d.devData.rho), star.u_floor, star.cooling_rho_limit);

    checkGpuErrors(hipDeviceSynchronize());
}

template void betaCoolingGPU(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, const StarData&);

template<typename Tu, typename Tdu>
struct AbsDivide
{
    HOST_DEVICE_FUN double operator()(const thrust::tuple<Tu, Tdu>& X)
    {
        return double{fabs(thrust::get<0>(X) / thrust::get<1>(X))};
    }
};

template<typename Dataset, typename StarData>
double duTimestepGPU(size_t first, size_t last, const Dataset& d, const StarData& star)
{
    cstone::LocalIndex numParticles = last - first;

    const auto* u  = rawPtr(d.devData.u);
    const auto* du = rawPtr(d.devData.du);

    using Tu  = std::decay_t<decltype(*u)>;
    using Tdu = std::decay_t<decltype(*du)>;

    auto begin = thrust::make_zip_iterator(u, du);
    auto end   = thrust::make_zip_iterator(u + numParticles, du + numParticles);

    double init = INFINITY;

    return star.K_u *
           thrust::transform_reduce(thrust::device, begin, end, AbsDivide<Tu, Tdu>{}, init, thrust::minimum<double>{});
}

template double duTimestepGPU(size_t, size_t, const sphexa::ParticlesData<cstone::GpuTag>&, const StarData&);

} // namespace disk
